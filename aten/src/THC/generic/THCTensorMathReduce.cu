
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMathReduce.cu"
#else

void THCTensor_(sum)(THCState* state, THCTensor *self, THCTensor *src, int dimension, int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  if (!THC_reduceDim<scalar_t>(state, self, src,
                           thrust::identity<accreal>{},
                           ReduceAdd<accreal>{},
                           thrust::identity<accreal>{},
                           scalar_cast<accreal>(0),
                           dimension,
                           keepdim)) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(prod)(THCState* state, THCTensor *self, THCTensor *src, int dimension, int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  if (!THC_reduceDim<scalar_t>(state, self, src,
                           thrust::identity<accreal>{},
                           ReduceMultiply<accreal>{},
                           thrust::identity<accreal>{},
                           scalar_cast<accreal>(1),
                           dimension,
                           keepdim)) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(mean)(THCState *state, THCTensor *self, THCTensor *src, int dim, int keepdim)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  const accreal size = scalar_cast<accreal>(THCTensor_(size)(state, src, dim));
  if (!THC_reduceDim<scalar_t>(state, self, src,
                           thrust::identity<accreal>{},
                           ReduceAdd<accreal>{},
                           ReduceDivide<accreal>{size},
                           scalar_cast<accreal>(0),
                           dim,
                           keepdim)) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

void THCTensor_(renorm)(THCState *state, THCTensor* self, THCTensor* src, scalar_t value, int dimension, scalar_t maxnorm)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  THCTensor *self_;
  THCTensor *src_ = THCTensor_(newTranspose)(state, src, dimension, 0);
  THCTensor *data = THCTensor_(newClone)(state, src_);
  int64_t numel = THCTensor_(nElement)(state, data);

  THArgCheck(dimension >= 0 && dimension < THCTensor_(nDimensionLegacyNoScalars)(state, src), 3, "invalid dimension");
  THArgCheck(THCNumerics<scalar_t>::gt(value, scalar_cast<scalar_t>(0)), 2, "non-positive-norm not supported");
  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, src) > 1, 1, "need at least 2 dimensions");

  if (numel > 0) {
    ptrdiff_t size = numel / THTensor_sizeLegacyNoScalars(data, 0);
    dim3 grid( THTensor_sizeLegacyNoScalars(data, 0));
    dim3 threads(32);

    THCTensor_kernel_renorm<scalar_t, accreal>
      <<<grid, threads, 0, THCState_getCurrentStream(state)>>>
      (THCTensor_(data)(state, data), scalar_cast<accreal>(value), size, scalar_cast<accreal>(maxnorm));

    cudaError_t errcode = cudaGetLastError();
    if(errcode != cudaSuccess)
      THError(cudaGetErrorString(errcode));
  }

  THCTensor_(free)(state, src_);
  self_ = THCTensor_(newTranspose)(state, data, dimension, 0);
  THCTensor_(resizeAs)(state, self, self_);
  THCTensor_(freeCopyTo)(state, self_, self);
  THCTensor_(free)(state, data);
}

void THCTensor_(std)(THCState *state, THCTensor *self_, THCTensor *src, int dimension, int biased, int keepdim)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));

  WelfordData<accreal, scalar_t> init;
  init.reset();
  if (!THC_reduceDim<scalar_t>(state, self_, src,
                           ModifyWelford<WelfordData<accreal, scalar_t>>{},
                           ReduceWelford<accreal, scalar_t>{},
                           VarianceWelford<accreal, scalar_t>{biased, true},
                           init,
                           dimension,
                           keepdim)) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(var)(THCState *state, THCTensor *self_, THCTensor *src, int dimension, int biased, int keepdim)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));

  WelfordData<accreal, scalar_t> init;
  init.reset();
  if (!THC_reduceDim<scalar_t>(state, self_, src,
                           ModifyWelford<WelfordData<accreal, scalar_t>>{},
                           ReduceWelford<accreal, scalar_t>{},
                           VarianceWelford<accreal, scalar_t>{biased, false},
                           init,
                           dimension,
                           keepdim)) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

accreal THCTensor_(stdall)(THCState *state, THCTensor *self, int biased)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  return THCNumerics<accreal>::sqrt((THCTensor_(varall)(state, self, biased)));
}

accreal THCTensor_(varall)(THCState *state, THCTensor *self, int biased)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal mean = THCTensor_(meanall)(state, self);

  accreal val;
  if (!THC_reduceAll<scalar_t>(state, self,
                           SquareFunctor<accreal>(mean),
                           ReduceAdd<accreal>(),
                           scalar_cast<accreal>(0),
                           &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  val = THCNumerics<accreal>::div(
    val,
    scalar_cast<accreal>(std::max<int64_t>(0, THCTensor_(nElement)(state, self) - (biased ? 0 : 1)))
  );

  THCudaCheck(cudaGetLastError());
  return val;
}

void THCTensor_(norm)(THCState *state, THCTensor* self, THCTensor* src, scalar_t _value, int dimension, int keepdim)
{
  const accreal value = scalar_cast<accreal>(_value);
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(0))) {
    THC_reduceDim<scalar_t>(state, self, src,
                        TensorNonZeroOp<accreal>{},
                        ReduceAdd<accreal>{},
                        thrust::identity<accreal>{},
                        scalar_cast<accreal>(0),
                        dimension, keepdim);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(1))) {
    THC_reduceDim<scalar_t>(state, self, src,
                        TensorNormOp<accreal, 1>{value},
                        ReduceAdd<accreal>{},
                        thrust::identity<accreal>{},
                        scalar_cast<accreal>(0),
                        dimension, keepdim);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(2))) {
    THC_reduceDim<scalar_t>(state, self, src,
                        TensorNormOp<accreal, 2>{value},
                        ReduceAdd<accreal>{},
                        ReducePow<accreal>{scalar_cast<accreal>(.5)},
                        scalar_cast<accreal>(0),
                        dimension, keepdim);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(INFINITY))) {
    THC_reduceDim<scalar_t>(state, self, src,
                        TensorNormOp<accreal, 1>{value},
                        ReduceMax<accreal>{},
                        thrust::identity<accreal>{},
                        scalar_cast<accreal>(0),
                        dimension, keepdim);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(-INFINITY))) {
    THC_reduceDim<scalar_t>(state, self, src,
                        TensorNormOp<accreal, 1>{value},
                        ReduceMin<accreal>{},
                        thrust::identity<accreal>{},
                        scalar_cast<accreal>(INFINITY),
                        dimension, keepdim);
  } else {
    THC_reduceDim<scalar_t>(state, self, src,
                        TensorNormOp<accreal, -1>{value},
                        ReduceAdd<accreal>{},
                        ReducePow<accreal>{THCNumerics<accreal>::cinv(value)},
                        scalar_cast<accreal>(0),
                        dimension, keepdim);
  }

  THCudaCheck(cudaGetLastError());
}

accreal THCTensor_(normall)(THCState *state, THCTensor *self, scalar_t _value)
{
  const accreal value = scalar_cast<accreal>(_value);
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal result;

  if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(0))) {
    THC_reduceAll<scalar_t>(state, self,
                        TensorNonZeroOp<accreal>{},
                        ReduceAdd<accreal>{},
                        scalar_cast<accreal>(0),
                        &result, 0);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(1))) {
    THC_reduceAll<scalar_t>(state, self,
                        TensorNormOp<accreal, 1>{value},
                        ReduceAdd<accreal>{},
                        scalar_cast<accreal>(0),
                        &result, 0);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(2))) {
    THC_reduceAll<scalar_t>(state, self,
                        TensorNormOp<accreal, 2>{value},
                        ReduceAdd<accreal>{},
                        scalar_cast<accreal>(0),
                        &result, 0);
    result = THCNumerics<accreal>::sqrt(result);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(INFINITY))) {
    THC_reduceAll<scalar_t>(state, self,
                        TensorNormOp<accreal, 1>{value},
                        ReduceMax<accreal>{},
                        scalar_cast<accreal>(0),
                        &result, 0);
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(-INFINITY))) {
    THC_reduceAll<scalar_t>(state, self,
                        TensorNormOp<accreal, 1>{value},
                        ReduceMin<accreal>{},
                        scalar_cast<accreal>(INFINITY),
                        &result, 0);
  } else {
    THC_reduceAll<scalar_t>(state, self,
                        TensorNormOp<accreal, -1>{value},
                        ReduceAdd<accreal>{},
                        scalar_cast<accreal>(0),
                        &result, 0);
    result = THCNumerics<accreal>::pow(result,
                                       THCNumerics<accreal>::cinv(value));
  }

  THCudaCheck(cudaGetLastError());
  return result;
}

accreal THCTensor_(dist)(THCState *state, THCTensor *self,
                         THCTensor *src, scalar_t _value)
{
  const accreal value = scalar_cast<accreal>(_value);
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  self = THCTensor_(newContiguous)(state, self);
  ptrdiff_t size = THCTensor_(nElement)(state, self);
  src = THCTensor_(newContiguous)(state, src);
  thrust::device_ptr<scalar_t> self_data(THCTensor_(data)(state, self));
  thrust::device_ptr<scalar_t> src_data(THCTensor_(data)(state, src));

  THCThrustAllocator thrustAlloc(state);
  accreal result;

  if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(INFINITY))) {
    result = thrust::inner_product(
#if CUDA_VERSION >= 7000
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
      self_data, self_data+size, src_data, scalar_cast<accreal>(0),
      ReduceMax<accreal>(),
      ThrustTensorDistOp<scalar_t, accreal>(scalar_cast<scalar_t>(1)));
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(-INFINITY))) {
    result = thrust::inner_product(
#if CUDA_VERSION >= 7000
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
      self_data, self_data+size, src_data, scalar_cast<accreal>(INFINITY),
      ReduceMin<accreal>(),
      ThrustTensorDistOp<scalar_t, accreal>(scalar_cast<scalar_t>(1)));
  } else if (THCNumerics<accreal>::eq(value, scalar_cast<accreal>(0))) {
    result = thrust::inner_product(
#if CUDA_VERSION >= 7000
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
      self_data, self_data+size, src_data, scalar_cast<accreal>(0),
      thrust::plus<accreal>(),
      ThrustTensorDistOp<scalar_t, accreal>(scalar_cast<scalar_t>(0)));
  } else {
    result = thrust::inner_product(
#if CUDA_VERSION >= 7000
      thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
      self_data, self_data+size, src_data, scalar_cast<accreal>(0),
      thrust::plus<accreal>(),
      ThrustTensorDistOp<scalar_t, accreal>(value));

    result = THCNumerics<accreal>::pow(result, THCNumerics<accreal>::cinv(value));
  }
  THCTensor_(free)(state, src);
  THCTensor_(free)(state, self);

  return result;
}

#endif

accreal THCTensor_(sumall)(THCState *state, THCTensor *self) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal val;
  if (!THC_reduceAll<scalar_t>(state, self,
                           thrust::identity<accreal>{},
                           ReduceAdd<accreal>{},
                           scalar_cast<accreal>(0),
                           &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
  return val;
}

accreal THCTensor_(prodall)(THCState *state, THCTensor *self) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal val;
  if (!THC_reduceAll<scalar_t>(state, self,
                           thrust::identity<accreal>{},
                           ReduceMultiply<accreal>{},
                           scalar_cast<accreal>(1),
                           &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
  return val;
}

accreal THCTensor_(meanall)(THCState *state, THCTensor *self)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  return THCTensor_(sumall)(state, self)/THCTensor_(nElement)(state, self);
}

scalar_t THCTensor_(minall)(THCState *state, THCTensor *self) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal val;
  if (!THC_reduceAll<scalar_t>(state, self,
                           thrust::identity<accreal>{},
                           ReduceMin<accreal>{},
                           THCNumerics<accreal>::upper_bound(), &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
  return scalar_cast<scalar_t>(val);
}

scalar_t THCTensor_(maxall)(THCState *state, THCTensor *self) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal val;
  if (!THC_reduceAll<scalar_t>(state, self,
                           thrust::identity<accreal>{},
                           ReduceMax<accreal>{},
                           THCNumerics<accreal>::lower_bound(), &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
  return scalar_cast<scalar_t>(val);
}

scalar_t THCTensor_(medianall)(THCState *state, THCTensor *self) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));

  scalar_t val;
  ptrdiff_t nelem, k;

  nelem = THCTensor_(nElement)(state, self);
  k = (nelem-1) >> 1;

  THCTensor *view = THCTensor_(newView)(state, self, {nelem});

  THCTensor *sorted = THCTensor_(new)(state);
  THCudaLongTensor *indices = THCudaLongTensor_new(state);

  THCTensor_(sort)(state, sorted, indices, view, 0, 0);

  val = THCTensor_(get1d)(state, sorted, k);

  THCTensor_(free)(state, view);
  THCTensor_(free)(state, sorted);
  THCudaLongTensor_free(state, indices);

  THCudaCheck(cudaGetLastError());

  return val;
}

void THCTensor_(median)(THCState *state,
                        THCTensor *values,
                        THCudaLongTensor *indices,
                        THCTensor *self,
                        int dimension,
                        int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));

  int64_t t_size_dim, k;

  t_size_dim = THCTensor_(size)(state, self, dimension);

  k = (t_size_dim-1) >> 1;

  THCTensor *sorted = THCTensor_(new)(state);
  THCudaLongTensor *sorted_indices = THCudaLongTensor_new(state);

  THCTensor_(sort)(state, sorted, sorted_indices, self, dimension, 0);

  THCTensor *newValues = THCTensor_(newNarrow)(state, sorted, dimension, k, 1);
  THCudaLongTensor *newIndices = THCudaLongTensor_newNarrow(state, sorted_indices, dimension, k, 1);

  THCTensor_(free)(state, sorted);
  THCudaLongTensor_free(state, sorted_indices);

  if (!keepdim) {
    THCTensor_(squeeze1d)(state, newValues, newValues, dimension);
    THCudaLongTensor_squeeze1d(state, newIndices, newIndices, dimension);
  }

  THCTensor_(resizeAs)(state, values, newValues);
  THCudaLongTensor_resizeAs(state, indices, newIndices);
  THCTensor_(copy)(state, values, newValues);
  THCudaLongTensor_copy(state, indices, newIndices);

  THCTensor_(free)(state, newValues);
  THCudaLongTensor_free(state, newIndices);

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(max)(THCState *state,
                     THCTensor *values,
                     THCudaLongTensor *indices,
                     THCTensor *src,
                     int dimension,
                     int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, values, indices, src));

  thrust::pair<scalar_t, int64_t>
    init =
    thrust::make_pair<scalar_t, int64_t>(
      THCNumerics<scalar_t>::lower_bound(), 0);

  return THC_reduceDimIndex<scalar_t, int64_t>(
    state, values, indices, src, dimension, keepdim, init,
    MaxValuePair<scalar_t, int64_t>());
}

void THCTensor_(min)(THCState *state,
                     THCTensor *values,
                     THCudaLongTensor *indices,
                     THCTensor *src,
                     int dimension,
                     int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, values, indices, src));

  thrust::pair<scalar_t, int64_t>
    init =
    thrust::make_pair<scalar_t, int64_t>(
      THCNumerics<scalar_t>::upper_bound(), 0);

  return THC_reduceDimIndex<scalar_t, int64_t>(
    state, values, indices, src, dimension, keepdim, init,
    MinValuePair<scalar_t, int64_t>());
}

#endif
